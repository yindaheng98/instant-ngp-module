#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <zstr.hpp>

#include <fstream>
#include <set>
#include <unordered_set>

#ifdef NGP_GUI
#	include <imgui/backends/imgui_impl_glfw.h>
#	include <imgui/backends/imgui_impl_opengl3.h>
#	include <imgui/imgui.h>
#	include <imguizmo/ImGuizmo.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#	include <GLFW/glfw3native.h>
#	include <cuda_gl_interop.h>

#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace std::literals::chrono_literals;

namespace ngp {
GPUMemory<bool> accu_grid_hit;
GPUMemory<bool> last_grid_hit;

void Testbed::do_grid_hit(GPUMemory<uint32_t>* grid_hit) {
    const uint64_t K = 64;
    uint64_t* counter_gpu;
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * K));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * K));
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), counter_gpu=counter_gpu, K=K] __device__ (size_t i) {
        for (uint64_t k=0;k<K;k++)
        if (grid_hit[i] > k) atomicAdd(counter_gpu + k, 1);
    });
    uint64_t counter_cpu[K];
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t) * K, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    // for (uint64_t k=0;k<K;k++)
    // tlog::info() << grid_hit->data() << ' ' << counter_cpu[k] << '/' << grid_hit->size();
    tlog::info() << grid_hit->data() << ' ' << counter_cpu[0] << '/' << grid_hit->size();

    if (accu_grid_hit.size() != grid_hit->size()) {
        accu_grid_hit.resize(grid_hit->size());
        accu_grid_hit.memset(0);
    }
    if (last_grid_hit.size() != grid_hit->size()) {
        last_grid_hit.resize(grid_hit->size());
        last_grid_hit.memset(0);
    }
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 2));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 2));
    uint64_t* accu_counter_gpu = counter_gpu;
    uint64_t* last_counter_gpu = counter_gpu + 1;
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data(), accu_counter_gpu, last_counter_gpu] __device__ (size_t i) {
        if (grid_hit[i] > 0 && !last_grid_hit[i]) atomicAdd(last_counter_gpu, 1);
        if (grid_hit[i] > 0 && !accu_grid_hit[i]) atomicAdd(accu_counter_gpu, 1);
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t) * 2, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << grid_hit->data() << ' ' << counter_cpu[0] << " not overlap accu" << ' ' << counter_cpu[1] << " not overlap last";

    if (last_grid_frame.size() != n_params() || this_grid_frame.size() != n_params()) return;
    size_t offset = n_params() - grid_hit->size();
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 3));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 3));
    uint64_t* inter_counter_gpu = counter_gpu;
    uint64_t* intra_counter_gpu = counter_gpu + 1;
	uint64_t* equal_counter_gpu = counter_gpu + 2;
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_frame=last_grid_frame.data() + offset, this_grid_frame=this_grid_frame.data() + offset, inter_counter_gpu, intra_counter_gpu, equal_counter_gpu] __device__ (size_t i) {
        if (grid_hit[i] <= 0) return;
        if (this_grid_frame[i] == last_grid_frame[i] + 1) atomicAdd(inter_counter_gpu, 1);
        else if (last_grid_frame[i] != this_grid_frame[i]) atomicAdd(intra_counter_gpu, 1);
		else atomicAdd(equal_counter_gpu, 1);
    });
    uint64_t int_counter_cpu[2];
    CUDA_CHECK_THROW(hipMemcpyAsync(int_counter_cpu, counter_gpu, sizeof(uint64_t) * 3, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "inter " << int_counter_cpu[0] << " intra " << int_counter_cpu[1] << " equal " << int_counter_cpu[2];

    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data()] __device__ (size_t i) {
        last_grid_hit[i] = grid_hit[i] > 0;
        accu_grid_hit[i] = grid_hit[i] > 0 || accu_grid_hit[i];
    });
}

}

