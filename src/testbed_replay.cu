#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <thrust/partition.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <zstr.hpp>

#include <fstream>
#include <set>
#include <unordered_set>

#ifdef NGP_GUI
#	include <imgui/backends/imgui_impl_glfw.h>
#	include <imgui/backends/imgui_impl_opengl3.h>
#	include <imgui/imgui.h>
#	include <imguizmo/ImGuizmo.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#	include <GLFW/glfw3native.h>
#	include <cuda_gl_interop.h>

#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace std::literals::chrono_literals;
#define MIN_RESIDUAL 0.01

namespace ngp {
GPUMemory<bool> accu_grid_hit;
GPUMemory<bool> last_grid_hit;
GPUMemory<network_precision_t> last_params;
GPUMemory<network_precision_t> inter_params;
GPUMemory<network_precision_t> intra_params;
GPUMemory<network_precision_t> residual_topk_i;
GPUMemory<network_precision_t> residual_topk_o;
unsigned int M = 10000;
int64_t the_frame = 0;
template< typename... Args >
std::string string_sprintf( const char* format, Args... args ) {
  int length = std::snprintf( nullptr, 0, format, args... );
  assert( length >= 0 );

  char* buf = new char[length + 1];
  std::snprintf( buf, length + 1, format, args... );

  std::string str( buf );
  delete[] buf;
  return str;
}
network_precision_t topk(network_precision_t* input, int length, int k) {
    thrust::sort(thrust::device, input, input+length, thrust::greater<network_precision_t>());
    network_precision_t top;
    CUDA_CHECK_THROW(hipMemcpy(&top, input+k, sizeof(network_precision_t), hipMemcpyDeviceToHost));
    return top;
}

void Testbed::do_grid_hit(GPUMemory<uint32_t>* grid_hit) {
    uint64_t* counter_gpu;
    uint64_t counter_cpu[32];

    // 统计：被调用超过k次的参数数量
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t)));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t)));
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), counter_gpu] __device__ (size_t i) {
        if (grid_hit[i] > 0) atomicAdd(counter_gpu, 1);
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t), hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "total " << counter_cpu[0] << '/' << grid_hit->size(); // 输出counter_cpu[0]是被调用过至少一次的参数数量

    if (accu_grid_hit.size() != grid_hit->size()) {
        accu_grid_hit.resize(grid_hit->size());
        accu_grid_hit.memset(0);
    }
    if (last_grid_hit.size() != grid_hit->size()) {
        last_grid_hit.resize(grid_hit->size());
        last_grid_hit.memset(0);
    }
    // 统计：当前视角和上一个视角有多少参数相交；当前视角和之前所有视角有多少参数相交
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 2));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 2));
    uint64_t* accu_counter_gpu = counter_gpu;
    uint64_t* last_counter_gpu = counter_gpu + 1;
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data(), accu_counter_gpu, last_counter_gpu] __device__ (size_t i) {
        if (grid_hit[i] > 0 && !last_grid_hit[i]) atomicAdd(last_counter_gpu, 1);
        if (grid_hit[i] > 0 && !accu_grid_hit[i]) atomicAdd(accu_counter_gpu, 1);
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t) * 2, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "static not overlap accu " << counter_cpu[0] << " not overlap last " << counter_cpu[1];

    if (last_params.size() != n_params()) { last_params.resize(n_params()); last_params.memset(0); }
    if (inter_params.size() != n_params()) inter_params.resize(n_params()); inter_params.memset(0);
    if (intra_params.size() != n_params()) intra_params.resize(n_params()); intra_params.memset(0);
    if (residual_topk_i.size() != grid_hit->size()) residual_topk_i.resize(grid_hit->size()); residual_topk_i.memset(0);
    if (residual_topk_o.size() != grid_hit->size()) residual_topk_o.resize(grid_hit->size()); residual_topk_o.memset(0);
    // 核心过程：过滤掉小残差
    // 统计：需要传完整参数的参数数量，过滤掉小残差后的残差数量和不变的参数数量
    size_t offset = n_params() - grid_hit->size();
    CUDA_CHECK_THROW(hipMemcpy(last_params.data(), m_network->params(), sizeof(network_precision_t) * offset, hipMemcpyDeviceToDevice)); // MLP参数不会变
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 3));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 3));
    uint64_t* inter_counter_gpu = counter_gpu;
    uint64_t* intra_counter_gpu = counter_gpu + 1;
	uint64_t* equal_counter_gpu = counter_gpu + 2;
    parallel_for_gpu(m_stream.get(), grid_hit->size(),
    [
        grid_hit=grid_hit->data(),
        accu_grid_hit=accu_grid_hit.data(),
        params=m_network->params() + offset,
        last_params=last_params.data() + offset,
        inter_params=inter_params.data() + offset,
        intra_params=intra_params.data() + offset,
        residual_topk_i=residual_topk_i.data(),
        inter_counter_gpu, intra_counter_gpu, equal_counter_gpu
    ] __device__ (size_t i) {
        if (grid_hit[i] <= 0) return;
        if (!accu_grid_hit[i]) {
            atomicAdd(intra_counter_gpu, 1);
            intra_params[i] = params[i];
            return;
        }
        network_precision_t residual = params[i] - last_params[i];
        if (residual > (network_precision_t)MIN_RESIDUAL || residual < -(network_precision_t)MIN_RESIDUAL) {
            residual_topk_i[atomicAdd(inter_counter_gpu, 1)] = residual;
            inter_params[i] = residual;
        }
		else {
            atomicAdd(equal_counter_gpu, 1);
        }
    });
    uint64_t int_counter_cpu[3];
    CUDA_CHECK_THROW(hipMemcpyAsync(int_counter_cpu, counter_gpu, sizeof(uint64_t) * 3, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    tlog::info() << "dynamic inter " << int_counter_cpu[0] << " intra " << int_counter_cpu[1] << " equal " << int_counter_cpu[2];

    // 核心过程：top k
    uint64_t inter_counter_cpu = int_counter_cpu[0];
    parallel_for_gpu(m_stream.get(), inter_counter_cpu, [input=residual_topk_i.data(), output=residual_topk_o.data()] __device__ (size_t i) {
        output[i] = (input[i]>=(network_precision_t)0)?input[i]:-input[i];
    });
    network_precision_t top = topk(residual_topk_o.data(), inter_counter_cpu, fminf(M, int_counter_cpu[0]));
    tlog::info() << "top " << fminf(M, int_counter_cpu[0]) << " = " << (float)top;

    if (inter_params.size() != n_params()) inter_params.resize(n_params()); inter_params.memset(0);
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 3));
    // 核心过程：k th 残差过滤
    // 统计：需要传完整参数的参数数量，过滤掉残差后的残差数量和不变的参数数量
    parallel_for_gpu(m_stream.get(), grid_hit->size(),
    [
        grid_hit=grid_hit->data(),
        accu_grid_hit=accu_grid_hit.data(),
        params=m_network->params() + offset,
        last_params=last_params.data() + offset,
        inter_params=inter_params.data() + offset,
        intra_params=intra_params.data() + offset,
        top, inter_counter_gpu, intra_counter_gpu, equal_counter_gpu
    ] __device__ (size_t i) {
        if (grid_hit[i] <= 0) return;
        if (!accu_grid_hit[i]) {
            atomicAdd(intra_counter_gpu, 1);
            return;
        }
        network_precision_t residual = params[i] - last_params[i];
        if (residual > top || residual < -top) {
            inter_params[i] = residual;
            atomicAdd(inter_counter_gpu, 1);
        }
		else {
            atomicAdd(equal_counter_gpu, 1);
        }
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(int_counter_cpu, counter_gpu, sizeof(uint64_t) * 3, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "filterd inter " << int_counter_cpu[0] << " intra " << int_counter_cpu[1] << " equal " << int_counter_cpu[2];

    // 核心过程：更新hit grid
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data()] __device__ (size_t i) {
        last_grid_hit[i] = grid_hit[i] > 0;
        accu_grid_hit[i] = grid_hit[i] > 0 || accu_grid_hit[i];
    });

    // 核心过程：模拟残差加
    parallel_for_gpu(m_stream.get(), grid_hit->size(),
    [
        params=m_network->params() + offset,
        last_params=last_params.data() + offset,
        inter_params=inter_params.data() + offset,
        intra_params=intra_params.data() + offset
    ] __device__ (size_t i) {
        if (intra_params[i] != (network_precision_t)0) last_params[i] = intra_params[i];
        else if (inter_params[i] != (network_precision_t)0) last_params[i] += inter_params[i];
        params[i] = last_params[i];
    });

    auto& snapshot = grid_hit_json;
    snapshot["params"] = last_params.size();
    snapshot["params_size"] = last_params;
    snapshot["density_grid_bitfield"] = m_nerf.density_grid_bitfield;
    snapshot["density_grid_bitfield_size"] = m_nerf.density_grid_bitfield.size();
    snapshot["density_grid_size"] = NERF_GRIDSIZE();
    GPUMemory<__half> density_grid_fp16(m_nerf.density_grid.size());
    parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
        density_grid_fp16[i] = (__half)density_grid[i];
    });

    fs::path save_path = native_string(string_sprintf(grid_hit_path.c_str(), the_frame));
    fs::create_directories(save_path.parent_path());
    save_grid_hit(save_path);
    the_frame++;
}

}

