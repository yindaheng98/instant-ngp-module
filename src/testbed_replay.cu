#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <zstr.hpp>

#include <fstream>
#include <set>
#include <unordered_set>

#ifdef NGP_GUI
#	include <imgui/backends/imgui_impl_glfw.h>
#	include <imgui/backends/imgui_impl_opengl3.h>
#	include <imgui/imgui.h>
#	include <imguizmo/ImGuizmo.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#	include <GLFW/glfw3native.h>
#	include <cuda_gl_interop.h>

#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace std::literals::chrono_literals;

namespace ngp {
GPUMemory<bool> accu_grid_hit;
GPUMemory<bool> last_grid_hit;
GPUMemory<int64_t> the_last_grid_frame;
GPUMemory<__half> the_params;
GPUMemory<__half> the_residuals;
int64_t the_frame = 0;
template< typename... Args >
std::string string_sprintf( const char* format, Args... args ) {
  int length = std::snprintf( nullptr, 0, format, args... );
  assert( length >= 0 );

  char* buf = new char[length + 1];
  std::snprintf( buf, length + 1, format, args... );

  std::string str( buf );
  delete[] buf;
  return str;
}

void Testbed::do_grid_hit(GPUMemory<uint32_t>* grid_hit) {
    const uint64_t K = 64;
    uint64_t* counter_gpu;
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * K));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * K));
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), counter_gpu, K] __device__ (size_t i) {
        for (uint64_t k=0;k<K;k++)
        if (grid_hit[i] > k) atomicAdd(counter_gpu + k, 1);
    });
    uint64_t counter_cpu[K];
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t) * K, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    // for (uint64_t k=0;k<K;k++)
    // tlog::info() << grid_hit->data() << ' ' << counter_cpu[k] << '/' << grid_hit->size();
    tlog::info() << "total " << counter_cpu[0] << '/' << grid_hit->size();

    if (accu_grid_hit.size() != grid_hit->size()) {
        accu_grid_hit.resize(grid_hit->size());
        accu_grid_hit.memset(0);
    }
    if (last_grid_hit.size() != grid_hit->size()) {
        last_grid_hit.resize(grid_hit->size());
        last_grid_hit.memset(0);
    }
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 2));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 2));
    uint64_t* accu_counter_gpu = counter_gpu;
    uint64_t* last_counter_gpu = counter_gpu + 1;
    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data(), accu_counter_gpu, last_counter_gpu] __device__ (size_t i) {
        if (grid_hit[i] > 0 && !last_grid_hit[i]) atomicAdd(last_counter_gpu, 1);
        if (grid_hit[i] > 0 && !accu_grid_hit[i]) atomicAdd(accu_counter_gpu, 1);
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(counter_cpu, counter_gpu, sizeof(uint64_t) * 2, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "static not overlap accu " << counter_cpu[0] << " not overlap last " << counter_cpu[1];

    if (last_grid_frame.size() != n_params() || this_grid_frame.size() != n_params() || current_residual.size() != n_params()) return;
    if (the_params.size() != n_params()) the_params.resize(n_params()); the_params.memset(0);
    if (the_residuals.size() != n_params()) the_residuals.resize(n_params()); the_residuals.memset(0);
    if (the_last_grid_frame.size() != grid_hit->size()) {
        the_last_grid_frame.resize(grid_hit->size());
        the_last_grid_frame.memset(-128);
    }
    size_t offset = n_params() - grid_hit->size();
    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 3));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 3));
    uint64_t* inter_counter_gpu = counter_gpu;
    uint64_t* intra_counter_gpu = counter_gpu + 1;
	uint64_t* equal_counter_gpu = counter_gpu + 2;
    parallel_for_gpu(m_stream.get(), grid_hit->size(),
    [
        grid_hit=grid_hit->data(),
        last_grid_frame=last_grid_frame.data() + offset,
        this_grid_frame=this_grid_frame.data() + offset,
        current_residual=current_residual.data() + offset,
        the_last_grid_frame=the_last_grid_frame.data(),
        the_params=the_params.data() + offset,
        the_residuals=the_residuals.data() + offset,
        inter_counter_gpu, intra_counter_gpu, equal_counter_gpu
    ] __device__ (size_t i) {
        if (grid_hit[i] <= 0) return;
        if (the_last_grid_frame[i] == this_grid_frame[i]) {
            atomicAdd(equal_counter_gpu, 1);
        }
        else if (the_last_grid_frame[i] == last_grid_frame[i]) {
            atomicAdd(inter_counter_gpu, 1);
            the_residuals[i] = current_residual[i];
        }
		else {
            atomicAdd(intra_counter_gpu, 1);
            the_params[i] = this_grid_frame[i];
        }
        the_last_grid_frame[i] = this_grid_frame[i];
    });
    uint64_t int_counter_cpu[3];
    CUDA_CHECK_THROW(hipMemcpyAsync(int_counter_cpu, counter_gpu, sizeof(uint64_t) * 3, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "dynamic inter " << int_counter_cpu[0] << " intra " << int_counter_cpu[1] << " equal " << int_counter_cpu[2];

    parallel_for_gpu(m_stream.get(), grid_hit->size(), [grid_hit=grid_hit->data(), last_grid_hit=last_grid_hit.data(), accu_grid_hit=accu_grid_hit.data()] __device__ (size_t i) {
        last_grid_hit[i] = grid_hit[i] > 0;
        accu_grid_hit[i] = grid_hit[i] > 0 || accu_grid_hit[i];
    });

    CUDA_CHECK_THROW(hipMalloc(&counter_gpu, sizeof(uint64_t) * 2));
    CUDA_CHECK_THROW(hipMemset(counter_gpu, 0, sizeof(uint64_t) * 2));
    inter_counter_gpu = counter_gpu;
    intra_counter_gpu = counter_gpu + 1;
    parallel_for_gpu(m_stream.get(), grid_hit->size(), 
    [
        grid_hit=grid_hit->data(),
        residual=current_residual.data() + offset,
        the_params=the_params.data() + offset,
        the_residuals=the_residuals.data() + offset,
        inter_counter_gpu, intra_counter_gpu, K
    ] __device__ (size_t i) {
        if ((float)the_params[i] != 0) atomicAdd(intra_counter_gpu, 1);
        if ((float)the_residuals[i] != 0) atomicAdd(inter_counter_gpu, 1);
    });
    CUDA_CHECK_THROW(hipMemcpyAsync(int_counter_cpu, counter_gpu, sizeof(uint64_t) * 2, hipMemcpyDeviceToHost, m_stream.get()));
    CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
    CUDA_CHECK_THROW(hipFree(counter_gpu));
    tlog::info() << "nonzero inter " << int_counter_cpu[0] << " intra " << int_counter_cpu[1];

    json data;
    data["intra"] = the_params;
    data["inter"] = the_residuals;
    fs::path path = native_string(string_sprintf(grid_hit_path.c_str(), the_frame));
    fs::create_directories(path.parent_path());
    std::ofstream f{path.str(), std::ios::out | std::ios::binary};
    zstr::ostream zf{f, zstr::default_buff_size, Z_BEST_COMPRESSION};
    json::to_bson(data, zf);
    the_frame++;
}

}

