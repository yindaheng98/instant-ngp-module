#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <zstr.hpp>

#include <fstream>
#include <set>
#include <unordered_set>

#ifdef NGP_GUI
#	include <imgui/backends/imgui_impl_glfw.h>
#	include <imgui/backends/imgui_impl_opengl3.h>
#	include <imgui/imgui.h>
#	include <imguizmo/ImGuizmo.h>
#	ifdef _WIN32
#		include <GL/gl3w.h>
#	else
#		include <GL/glew.h>
#	endif
#	include <GLFW/glfw3.h>
#	include <GLFW/glfw3native.h>
#	include <cuda_gl_interop.h>

#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace std::literals::chrono_literals;

namespace ngp {

int do_system(const std::string& cmd) {
#ifdef _WIN32
	tlog::info() << "> " << cmd;
	return _wsystem(utf8_to_utf16(cmd).c_str());
#else
	tlog::info() << "$ " << cmd;
	return system(cmd.c_str());
#endif
}
	
std::atomic<size_t> g_total_n_bytes_allocated{0};

json merge_parent_network_config(const json& child, const fs::path& child_path) {
	if (!child.contains("parent")) {
		return child;
	}
	fs::path parent_path = child_path.parent_path() / std::string(child["parent"]);
	tlog::info() << "Loading parent network config from: " << parent_path.str();
	std::ifstream f{native_string(parent_path)};
	json parent = json::parse(f, nullptr, true, true);
	parent = merge_parent_network_config(parent, parent_path);
	parent.merge_patch(child);
	return parent;
}

std::string get_filename_in_data_path_with_suffix(fs::path data_path, fs::path network_config_path, const char* suffix) {
	// use the network config name along with the data path to build a filename with the requested suffix & extension
	std::string default_name = network_config_path.basename();
	if (default_name == "") {
		default_name = "base";
	}

	if (data_path.empty()) {
		return default_name + std::string(suffix);
	}

	if (data_path.is_directory()) {
		return (data_path / (default_name + std::string{suffix})).str();
	}

	return data_path.stem().str() + "_" + default_name + std::string(suffix);
}

void Testbed::update_imgui_paths() {
	snprintf(m_imgui.cam_path_path, sizeof(m_imgui.cam_path_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_cam.json").c_str());
	snprintf(m_imgui.extrinsics_path, sizeof(m_imgui.extrinsics_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_extrinsics.json").c_str());
	snprintf(m_imgui.mesh_path, sizeof(m_imgui.mesh_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".obj").c_str());
	snprintf(m_imgui.snapshot_path, sizeof(m_imgui.snapshot_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".ingp").c_str());
	snprintf(m_imgui.video_path, sizeof(m_imgui.video_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_video.mp4").c_str());
}

void Testbed::load_training_data(const fs::path& path) {
	if (!path.exists()) {
		throw std::runtime_error{fmt::format("Data path '{}' does not exist.", path.str())};
	}

	// Automatically determine the mode from the first scene that's loaded
	ETestbedMode scene_mode = mode_from_scene(path.str());
	if (scene_mode == ETestbedMode::None) {
		throw std::runtime_error{fmt::format("Unknown scene format for path '{}'.", path.str())};
	}

	set_mode(scene_mode);

	m_data_path = path;

	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:   load_nerf(path); break;
		case ETestbedMode::Sdf:    load_mesh(path); break;
		case ETestbedMode::Image:  load_image(path); break;
		case ETestbedMode::Volume: load_volume(path); break;
		default: throw std::runtime_error{"Invalid testbed mode."};
	}

	m_training_data_available = true;

	update_imgui_paths();
}

void Testbed::reload_training_data() {
	if (m_data_path.exists()) {
		load_training_data(m_data_path.str());
	}
}

void Testbed::clear_training_data() {
	m_training_data_available = false;
	m_nerf.training.dataset.metadata.clear();
}

fs::path Testbed::find_network_config(const fs::path& network_config_path) {
	if (network_config_path.exists()) {
		return network_config_path;
	}

	// The following resolution steps do not work if the path is absolute. Treat it as nonexistent.
	if (network_config_path.is_absolute()) {
		return network_config_path;
	}

	fs::path candidate = root_dir()/"configs"/to_string(m_testbed_mode)/network_config_path;
	if (candidate.exists()) {
		return candidate;
	}

	return network_config_path;
}

json Testbed::load_network_config(std::istream& stream, bool is_compressed) {
	if (is_compressed) {
		zstr::istream zstream{stream};
		return json::from_msgpack(zstream);
	}
	return json::from_bson(stream);
}

json Testbed::load_network_config(const fs::path& network_config_path) {
	bool is_snapshot = equals_case_insensitive(network_config_path.extension(), "bson") || equals_case_insensitive(network_config_path.extension(), "ingp");
	if (network_config_path.empty() || !network_config_path.exists()) {
		throw std::runtime_error{fmt::format("Network {} '{}' does not exist.", is_snapshot ? "snapshot" : "config", network_config_path.str())};
	}

	tlog::info() << "Loading network " << (is_snapshot ? "snapshot" : "config") << " from: " << network_config_path;

	json result;
	if (is_snapshot) {
		std::ifstream f{native_string(network_config_path), std::ios::in | std::ios::binary};
		if (equals_case_insensitive(network_config_path.extension(), "ingp")) {
			// zstr::ifstream applies zlib compression.
			zstr::istream zf{f};
			result = json::from_msgpack(zf);
		} else if (equals_case_insensitive(network_config_path.extension(), "bson")) {
			result = json::from_bson(f);
		} else {
			result = json::from_msgpack(f);
		}
		// we assume parent pointers are already resolved in snapshots.
	} else if (equals_case_insensitive(network_config_path.extension(), "json")) {
		std::ifstream f{native_string(network_config_path)};
		result = json::parse(f, nullptr, true, true);
		result = merge_parent_network_config(result, network_config_path);
	}

	return result;
}

void Testbed::reload_network_from_file(const fs::path& path) {
	if (!path.empty()) {
		fs::path candidate = find_network_config(path);
		if (candidate.exists() || !m_network_config_path.exists()) {
			// Store the path _argument_ in the member variable. E.g. for the base config,
			// it'll store `base.json`, even though the loaded config will be
			// config/<mode>/base.json. This has the benefit of switching to the
			// appropriate config when switching modes.
			m_network_config_path = path;
		}
	}

	// If the testbed mode hasn't been decided yet, don't load a network yet, but
	// still keep track of the requested config (see above).
	if (m_testbed_mode == ETestbedMode::None) {
		return;
	}

	fs::path full_network_config_path = find_network_config(m_network_config_path);
	bool is_snapshot = equals_case_insensitive(full_network_config_path.extension(), "bson");

	if (!full_network_config_path.exists()) {
		tlog::warning() << "Network " << (is_snapshot ? "snapshot" : "config") << " path '" << full_network_config_path << "' does not exist.";
	} else {
		m_network_config = load_network_config(full_network_config_path);
	}

	// Reset training if we haven't loaded a snapshot of an already trained model, in which case, presumably the network
	// configuration changed and the user is interested in seeing how it trains from scratch.
	if (!is_snapshot) {
		reset_network();
	}
}

void Testbed::reload_network_from_json(const json& json, const std::string& config_base_path) {
	// config_base_path is needed so that if the passed in json uses the 'parent' feature, we know where to look...
	// be sure to use a filename, or if a directory, end with a trailing slash
	m_network_config = merge_parent_network_config(json, config_base_path);
	reset_network();
}

void Testbed::load_file(const fs::path& path) {
	if (!path.exists()) {
		// If the path doesn't exist, but a network config can be resolved, load that.
		if (equals_case_insensitive(path.extension(), "json") && find_network_config(path).exists()) {
			reload_network_from_file(path);
			return;
		}

		tlog::error() << "File '" << path.str() << "' does not exist.";
		return;
	}

	if (equals_case_insensitive(path.extension(), "ingp") || equals_case_insensitive(path.extension(), "bson")) {
		load_snapshot(path);
		return;
	}

	// If we get a json file, we need to parse it to determine its purpose.
	if (equals_case_insensitive(path.extension(), "json")) {
		json file;
		{
			std::ifstream f{native_string(path)};
			file = json::parse(f, nullptr, true, true);
		}

		// Snapshot in json format... inefficient, but technically supported.
		if (file.contains("snapshot")) {
			load_snapshot(path);
			return;
		}

		// Regular network config
		if (file.contains("parent") || file.contains("network") || file.contains("encoding") || file.contains("loss") || file.contains("optimizer")) {
			reload_network_from_file(path);
			return;
		}

		// Camera path
		if (file.contains("path")) {
			load_camera_path(path);
			return;
		}
	}

	// If the dragged file isn't any of the above, assume that it's training data
	try {
		bool was_training_data_available = m_training_data_available;
		load_training_data(path);

		if (!was_training_data_available) {
			// If we previously didn't have any training data and only now dragged
			// some into the window, it is very unlikely that the user doesn't
			// want to immediately start training on that data. So: go for it.
			m_train = true;
		}
	} catch (const std::runtime_error& e) {
		tlog::error() << "Failed to load training data: " << e.what();
	}
}

fs::path Testbed::root_dir() {
	if (m_root_dir.empty()) {
		set_root_dir(discover_root_dir());
	}

	return m_root_dir;
}

void Testbed::set_root_dir(const fs::path& dir) {
	m_root_dir = dir;
}

template <typename T>
void Testbed::dump_parameters_as_images(const T* params, const std::string& filename_base) {
	if (!m_network) {
		return;
	}

	size_t non_layer_params_width = 2048;

	size_t layer_params = 0;
	for (auto size : m_network->layer_sizes()) {
		layer_params += size.first * size.second;
	}

	size_t n_params = m_network->n_params();
	size_t n_non_layer_params = n_params - layer_params;

	std::vector<T> params_cpu_network_precision(layer_params + next_multiple(n_non_layer_params, non_layer_params_width));
	std::vector<float> params_cpu(params_cpu_network_precision.size(), 0.0f);
	CUDA_CHECK_THROW(hipMemcpy(params_cpu_network_precision.data(), params, n_params * sizeof(T), hipMemcpyDeviceToHost));

	for (size_t i = 0; i < n_params; ++i) {
		params_cpu[i] = (float)params_cpu_network_precision[i];
	}

	size_t offset = 0;
	size_t layer_id = 0;
	for (auto size : m_network->layer_sizes()) {
		save_exr(params_cpu.data() + offset, size.second, size.first, 1, 1, fmt::format("{}-layer-{}.exr", filename_base, layer_id).c_str());
		offset += size.first * size.second;
		++layer_id;
	}

	if (n_non_layer_params > 0) {
		std::string filename = fmt::format("{}-non-layer.exr", filename_base);
		save_exr(params_cpu.data() + offset, non_layer_params_width, n_non_layer_params / non_layer_params_width, 1, 1, filename.c_str());
	}
}

template void Testbed::dump_parameters_as_images<__half>(const __half*, const std::string&);
template void Testbed::dump_parameters_as_images<float>(const float*, const std::string&);

fs::path Testbed::training_data_path() const {
	return m_data_path.with_extension("training");
}

size_t Testbed::n_params() {
	return m_network ? m_network->n_params() : 0;
}

size_t Testbed::n_encoding_params() {
	return n_params() - first_encoder_param();
}

size_t Testbed::first_encoder_param() {
	if (!m_network) {
		return 0;
	}

	auto layer_sizes = m_network->layer_sizes();
	size_t first_encoder = 0;
	for (auto size : layer_sizes) {
		first_encoder += size.first * size.second;
	}

	return first_encoder;
}

uint32_t Testbed::network_width(uint32_t layer) const {
	return m_network ? m_network->width(layer) : 0;
}

uint32_t Testbed::network_num_forward_activations() const {
	return m_network ? m_network->num_forward_activations() : 0;
}

ELossType Testbed::string_to_loss_type(const std::string& str) {
	if (equals_case_insensitive(str, "L2")) {
		return ELossType::L2;
	} else if (equals_case_insensitive(str, "RelativeL2")) {
		return ELossType::RelativeL2;
	} else if (equals_case_insensitive(str, "L1")) {
		return ELossType::L1;
	} else if (equals_case_insensitive(str, "Mape")) {
		return ELossType::Mape;
	} else if (equals_case_insensitive(str, "Smape")) {
		return ELossType::Smape;
	} else if (equals_case_insensitive(str, "Huber") || equals_case_insensitive(str, "SmoothL1")) {
		// Legacy: we used to refer to the Huber loss (L2 near zero, L1 further away) as "SmoothL1".
		return ELossType::Huber;
	} else if (equals_case_insensitive(str, "LogL1")) {
		return ELossType::LogL1;
	} else {
		throw std::runtime_error{"Unknown loss type."};
	}
}

Testbed::NetworkDims Testbed::network_dims() const {
	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:   return network_dims_nerf(); break;
		case ETestbedMode::Sdf:    return network_dims_sdf(); break;
		case ETestbedMode::Image:  return network_dims_image(); break;
		case ETestbedMode::Volume: return network_dims_volume(); break;
		default: throw std::runtime_error{"Invalid mode."};
	}
}

void Testbed::reset_network(bool clear_density_grid) {
	m_sdf.iou_decay = 0;

	m_rng = default_rng_t{m_seed};

	// Start with a low rendering resolution and gradually ramp up
	m_render_ms.set(10000);

	// reset_accumulation();
	m_nerf.training.counters_rgb.rays_per_batch = 1 << 12;
	m_nerf.training.counters_rgb.measured_batch_size_before_compaction = 0;
	m_nerf.training.n_steps_since_cam_update = 0;
	m_nerf.training.n_steps_since_error_map_update = 0;
	m_nerf.training.n_rays_since_error_map_update = 0;
	m_nerf.training.n_steps_between_error_map_updates = 128;
	m_nerf.training.error_map.is_cdf_valid = false;
	m_nerf.training.density_grid_rng = default_rng_t{m_rng.next_uint()};

	m_nerf.training.reset_camera_extrinsics();

	if (clear_density_grid) {
		m_nerf.density_grid.memset(0);
		m_nerf.density_grid_bitfield.memset(0);

		set_all_devices_dirty();
	}

	m_loss_graph_samples = 0;

	// Default config
	json config = m_network_config;

	json& encoding_config = config["encoding"];
	json& loss_config = config["loss"];
	json& optimizer_config = config["optimizer"];
	json& network_config = config["network"];

	// If the network config is incomplete, avoid doing further work.
	/*
	if (config.is_null() || encoding_config.is_null() || loss_config.is_null() || optimizer_config.is_null() || network_config.is_null()) {
		return;
	}
	*/

	auto dims = network_dims();

	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.loss_type = string_to_loss_type(loss_config.value("otype", "L2"));

		// Some of the Nerf-supported losses are not supported by Loss,
		// so just create a dummy L2 loss there. The NeRF code path will bypass
		// the Loss in any case.
		loss_config["otype"] = "L2";
	}

	// Automatically determine certain parameters if we're dealing with the (hash)grid encoding
	if (to_lower(encoding_config.value("otype", "OneBlob")).find("grid") != std::string::npos) {
		encoding_config["n_pos_dims"] = dims.n_pos;

		m_n_features_per_level = encoding_config.value("n_features_per_level", 2u);

		if (encoding_config.contains("n_features") && encoding_config["n_features"] > 0) {
			m_n_levels = (uint32_t)encoding_config["n_features"] / m_n_features_per_level;
		} else {
			m_n_levels = encoding_config.value("n_levels", 16u);
		}

		m_level_stats.resize(m_n_levels);
		m_first_layer_column_stats.resize(m_n_levels);

		const uint32_t log2_hashmap_size = encoding_config.value("log2_hashmap_size", 15);

		m_base_grid_resolution = encoding_config.value("base_resolution", 0);
		if (!m_base_grid_resolution) {
			m_base_grid_resolution = 1u << ((log2_hashmap_size) / dims.n_pos);
			encoding_config["base_resolution"] = m_base_grid_resolution;
		}

		float desired_resolution = 2048.0f; // Desired resolution of the finest hashgrid level over the unit cube
		if (m_testbed_mode == ETestbedMode::Image) {
			desired_resolution = max(m_image.resolution) / 2.0f;
		} else if (m_testbed_mode == ETestbedMode::Volume) {
			desired_resolution = m_volume.world2index_scale;
		}

		// Automatically determine suitable per_level_scale
		m_per_level_scale = encoding_config.value("per_level_scale", 0.0f);
		if (m_per_level_scale <= 0.0f && m_n_levels > 1) {
			m_per_level_scale = std::exp(std::log(desired_resolution * (float)m_nerf.training.dataset.aabb_scale / (float)m_base_grid_resolution) / (m_n_levels-1));
			encoding_config["per_level_scale"] = m_per_level_scale;
		}

		tlog::info()
			<< "GridEncoding: "
			<< " Nmin=" << m_base_grid_resolution
			<< " b=" << m_per_level_scale
			<< " F=" << m_n_features_per_level
			<< " T=2^" << log2_hashmap_size
			<< " L=" << m_n_levels
			;
	}

	m_loss.reset(create_loss<network_precision_t>(loss_config));
	m_optimizer.reset(create_optimizer<network_precision_t>(optimizer_config));

	size_t n_encoding_params = 0;
	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.cam_exposure.resize(m_nerf.training.dataset.n_images, AdamOptimizer<vec3>(1e-3f));
		m_nerf.training.cam_pos_offset.resize(m_nerf.training.dataset.n_images, AdamOptimizer<vec3>(1e-4f));
		m_nerf.training.cam_rot_offset.resize(m_nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
		m_nerf.training.cam_focal_length_offset = AdamOptimizer<vec2>(1e-5f);

		m_nerf.reset_extra_dims(m_rng);

		json& dir_encoding_config = config["dir_encoding"];
		json& rgb_network_config = config["rgb_network"];

		uint32_t n_dir_dims = 3;
		uint32_t n_extra_dims = m_nerf.training.dataset.n_extra_dims();

		// Instantiate an additional model for each auxiliary GPU
		for (auto& device : m_devices) {
			device.set_nerf_network(std::make_shared<NerfNetwork<network_precision_t>>(
				dims.n_pos,
				n_dir_dims,
				n_extra_dims,
				dims.n_pos + 1, // The offset of 1 comes from the dt member variable of NerfCoordinate. HACKY
				encoding_config,
				dir_encoding_config,
				network_config,
				rgb_network_config
			));
		}

		m_network = m_nerf_network = primary_device().nerf_network();

		m_encoding = m_nerf_network->pos_encoding();
		n_encoding_params = m_encoding->n_params() + m_nerf_network->dir_encoding()->n_params();

		tlog::info()
			<< "Density model: " << dims.n_pos
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_nerf_network->pos_encoding()->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 1
			;

		tlog::info()
			<< "Color model:   " << n_dir_dims
			<< "--[" << std::string(dir_encoding_config["otype"])
			<< "]-->" << m_nerf_network->dir_encoding()->padded_output_width() << "+" << network_config.value("n_output_dims", 16u)
			<< "--[" << std::string(rgb_network_config["otype"])
			<< "(neurons=" << (int)rgb_network_config["n_neurons"] << ",layers=" << ((int)rgb_network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 3
			;

		// Create distortion map model
		{
			json& distortion_map_optimizer_config =  config.contains("distortion_map") && config["distortion_map"].contains("optimizer") ? config["distortion_map"]["optimizer"] : optimizer_config;

			m_distortion.resolution = ivec2(32);
			if (config.contains("distortion_map") && config["distortion_map"].contains("resolution")) {
				from_json(config["distortion_map"]["resolution"], m_distortion.resolution);
			}
			m_distortion.map = std::make_shared<TrainableBuffer<2, 2, float>>(m_distortion.resolution);
			m_distortion.optimizer.reset(create_optimizer<float>(distortion_map_optimizer_config));
			m_distortion.trainer = std::make_shared<Trainer<float, float>>(m_distortion.map, m_distortion.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(loss_config)}, m_seed);
		}
	} else {
		uint32_t alignment = network_config.contains("otype") && (equals_case_insensitive(network_config["otype"], "FullyFusedMLP") || equals_case_insensitive(network_config["otype"], "MegakernelMLP")) ? 16u : 8u;

		if (encoding_config.contains("otype") && equals_case_insensitive(encoding_config["otype"], "Takikawa")) {
			if (m_sdf.octree_depth_target == 0) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}

			if (!m_sdf.triangle_octree || m_sdf.triangle_octree->depth() != m_sdf.octree_depth_target) {
				m_sdf.triangle_octree.reset(new TriangleOctree{});
				m_sdf.triangle_octree->build(*m_sdf.triangle_bvh, m_sdf.triangles_cpu, m_sdf.octree_depth_target);
				m_sdf.octree_depth_target = m_sdf.triangle_octree->depth();
				m_sdf.brick_data.free_memory();
			}

			m_encoding.reset(new TakikawaEncoding<network_precision_t>(
				encoding_config["starting_level"],
				m_sdf.triangle_octree,
				string_to_interpolation_type(encoding_config.value("interpolation", "linear"))
			));

			m_sdf.uses_takikawa_encoding = true;
		} else {
			m_encoding.reset(create_encoding<network_precision_t>(dims.n_input, encoding_config));

			m_sdf.uses_takikawa_encoding = false;
			if (m_sdf.octree_depth_target == 0 && encoding_config.contains("n_levels")) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}
		}

		for (auto& device : m_devices) {
			device.set_network(std::make_shared<NetworkWithInputEncoding<network_precision_t>>(m_encoding, dims.n_output, network_config));
		}

		m_network = primary_device().network();

		n_encoding_params = m_encoding->n_params();

		tlog::info()
			<< "Model:         " << dims.n_input
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_encoding->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << dims.n_output
			;
	}

	size_t n_network_params = m_network->n_params() - n_encoding_params;

	tlog::info() << "  total_encoding_params=" << n_encoding_params << " total_network_params=" << n_network_params;

	m_trainer = std::make_shared<Trainer<float, network_precision_t, network_precision_t>>(m_network, m_optimizer, m_loss, m_seed);
	m_training_step = 0;
	m_training_start_time_point = std::chrono::steady_clock::now();

	// Create envmap model
	{
		json& envmap_loss_config = config.contains("envmap") && config["envmap"].contains("loss") ? config["envmap"]["loss"] : loss_config;
		json& envmap_optimizer_config =  config.contains("envmap") && config["envmap"].contains("optimizer") ? config["envmap"]["optimizer"] : optimizer_config;

		m_envmap.loss_type = string_to_loss_type(envmap_loss_config.value("otype", "L2"));

		m_envmap.resolution = m_nerf.training.dataset.envmap_resolution;
		m_envmap.envmap = std::make_shared<TrainableBuffer<4, 2, float>>(m_envmap.resolution);
		m_envmap.optimizer.reset(create_optimizer<float>(envmap_optimizer_config));
		m_envmap.trainer = std::make_shared<Trainer<float, float, float>>(m_envmap.envmap, m_envmap.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(envmap_loss_config)}, m_seed);

		if (m_nerf.training.dataset.envmap_data.data()) {
			m_envmap.trainer->set_params_full_precision(m_nerf.training.dataset.envmap_data.data(), m_nerf.training.dataset.envmap_data.size());
		}
	}

	set_all_devices_dirty();
}

bool Testbed::clear_tmp_dir() {
	wait_all(m_render_futures);
	m_render_futures.clear();

	bool success = true;
	auto tmp_dir = fs::path{"tmp"};
	if (tmp_dir.exists()) {
		if (tmp_dir.is_directory()) {
			for (const auto& path : fs::directory{tmp_dir}) {
				if (path.is_file()) {
					success &= path.remove_file();
				}
			}
		}

		success &= tmp_dir.remove_file();
	}

	return success;
}

// Increment this number when making a change to the snapshot format
static const size_t SNAPSHOT_FORMAT_VERSION = 1;

void Testbed::save_snapshot(const fs::path& path, bool include_optimizer_state, bool compress) {
	m_network_config["snapshot"] = m_trainer->serialize(include_optimizer_state);

	auto& snapshot = m_network_config["snapshot"];
	snapshot["version"] = SNAPSHOT_FORMAT_VERSION;
	snapshot["mode"] = to_string(m_testbed_mode);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["density_grid_size"] = NERF_GRIDSIZE();

		GPUMemory<__half> density_grid_fp16(m_nerf.density_grid.size());
		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid_fp16[i] = (__half)density_grid[i];
		});

		snapshot["density_grid_binary"] = density_grid_fp16;
		snapshot["nerf"]["aabb_scale"] = m_nerf.training.dataset.aabb_scale;

		snapshot["nerf"]["cam_pos_offset"] = m_nerf.training.cam_pos_offset;
		snapshot["nerf"]["cam_rot_offset"] = m_nerf.training.cam_rot_offset;
		snapshot["nerf"]["extra_dims_opt"] = m_nerf.training.extra_dims_opt;
	}

	snapshot["training_step"] = m_training_step;
	snapshot["loss"] = m_loss_scalar.val();
	snapshot["aabb"] = m_aabb;
	snapshot["bounding_radius"] = m_bounding_radius;
	snapshot["render_aabb_to_local"] = m_render_aabb_to_local;
	snapshot["render_aabb"] = m_render_aabb;
	snapshot["up_dir"] = m_up_dir;
	snapshot["sun_dir"] = m_sun_dir;
	snapshot["exposure"] = m_exposure;
	snapshot["background_color"] = m_background_color;

	snapshot["camera"]["matrix"] = m_camera;
	snapshot["camera"]["fov_axis"] = m_fov_axis;
	snapshot["camera"]["relative_focal_length"] = m_relative_focal_length;
	snapshot["camera"]["screen_center"] = m_screen_center;
	snapshot["camera"]["zoom"] = m_zoom;
	snapshot["camera"]["scale"] = m_scale;

	snapshot["camera"]["aperture_size"] = m_aperture_size;
	snapshot["camera"]["autofocus"] = m_autofocus;
	snapshot["camera"]["autofocus_target"] = m_autofocus_target;
	snapshot["camera"]["autofocus_depth"] = m_slice_plane_z;

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["nerf"]["rgb"]["rays_per_batch"] = m_nerf.training.counters_rgb.rays_per_batch;
		snapshot["nerf"]["rgb"]["measured_batch_size"] = m_nerf.training.counters_rgb.measured_batch_size;
		snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"] = m_nerf.training.counters_rgb.measured_batch_size_before_compaction;
		snapshot["nerf"]["dataset"] = m_nerf.training.dataset;
	}

	m_network_config_path = path;
	std::ofstream f{native_string(m_network_config_path), std::ios::out | std::ios::binary};
	if (equals_case_insensitive(m_network_config_path.extension(), "ingp")) {
		// zstr::ofstream applies zlib compression.
		zstr::ostream zf{f, zstr::default_buff_size, compress ? Z_DEFAULT_COMPRESSION : Z_NO_COMPRESSION};
		json::to_msgpack(m_network_config, zf);
	} else if (equals_case_insensitive(m_network_config_path.extension(), "bson")) {
		json::to_bson(m_network_config, f);
	} else {
		json::to_msgpack(m_network_config, f);
	}

	tlog::success() << "Saved snapshot '" << path.str() << "'";
}

void Testbed::load_snapshot(nlohmann::json config) {
	const auto& snapshot = config["snapshot"];
	if (snapshot.value("version", 0) < SNAPSHOT_FORMAT_VERSION) {
		throw std::runtime_error{"Snapshot uses an old format and can not be loaded."};
	}

	if (snapshot.contains("mode")) {
		set_mode(mode_from_string(snapshot["mode"]));
	} else if (snapshot.contains("nerf")) {
		// To be able to load old NeRF snapshots that don't specify their mode yet
		set_mode(ETestbedMode::Nerf);
	} else if (m_testbed_mode == ETestbedMode::None) {
		throw std::runtime_error{"Unknown snapshot mode. Snapshot must be regenerated with a new version of instant-ngp."};
	}

	m_aabb = snapshot.value("aabb", m_aabb);
	m_bounding_radius = snapshot.value("bounding_radius", m_bounding_radius);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		if (snapshot["density_grid_size"] != NERF_GRIDSIZE()) {
			throw std::runtime_error{"Incompatible grid size."};
		}

		m_nerf.training.counters_rgb.rays_per_batch = snapshot["nerf"]["rgb"]["rays_per_batch"];
		m_nerf.training.counters_rgb.measured_batch_size = snapshot["nerf"]["rgb"]["measured_batch_size"];
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"];

		// If we haven't got a nerf dataset loaded, load dataset metadata from the snapshot
		// and render using just that.
		if (m_data_path.empty() && snapshot["nerf"].contains("dataset")) {
			m_nerf.training.dataset = snapshot["nerf"]["dataset"];
			load_nerf(m_data_path);
		} else {
			if (snapshot["nerf"].contains("aabb_scale")) {
				m_nerf.training.dataset.aabb_scale = snapshot["nerf"]["aabb_scale"];
			}

			if (snapshot["nerf"].contains("dataset")) {
				m_nerf.training.dataset.n_extra_learnable_dims = snapshot["nerf"]["dataset"].value("n_extra_learnable_dims", m_nerf.training.dataset.n_extra_learnable_dims);
			}
		}

		load_nerf_post();

		GPUMemory<__half> density_grid_fp16 = snapshot["density_grid_binary"];
		m_nerf.density_grid.resize(density_grid_fp16.size());

		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid[i] = (float)density_grid_fp16[i];
		});

		if (m_nerf.density_grid.size() == NERF_GRID_N_CELLS() * (m_nerf.max_cascade + 1)) {
			update_density_grid_mean_and_bitfield(nullptr);
		} else if (m_nerf.density_grid.size() != 0) {
			// A size of 0 indicates that the density grid was never populated, which is a valid state of a (yet) untrained model.
			throw std::runtime_error{"Incompatible number of grid cascades."};
		}
	}

	// Needs to happen after `load_nerf_post()`
	m_sun_dir = snapshot.value("sun_dir", m_sun_dir);
	m_exposure = snapshot.value("exposure", m_exposure);

#ifdef NGP_GUI
	if (!m_hmd)
#endif
	m_background_color = snapshot.value("background_color", m_background_color);

	if (snapshot.contains("camera")) {
		m_camera = snapshot["camera"].value("matrix", m_camera);
		m_fov_axis = snapshot["camera"].value("fov_axis", m_fov_axis);
		if (snapshot["camera"].contains("relative_focal_length")) from_json(snapshot["camera"]["relative_focal_length"], m_relative_focal_length);
		if (snapshot["camera"].contains("screen_center")) from_json(snapshot["camera"]["screen_center"], m_screen_center);
		m_zoom = snapshot["camera"].value("zoom", m_zoom);
		m_scale = snapshot["camera"].value("scale", m_scale);

		m_aperture_size = snapshot["camera"].value("aperture_size", m_aperture_size);
		if (m_aperture_size != 0) {
			m_dlss = false;
		}

		m_autofocus = snapshot["camera"].value("autofocus", m_autofocus);
		if (snapshot["camera"].contains("autofocus_target")) from_json(snapshot["camera"]["autofocus_target"], m_autofocus_target);
		m_slice_plane_z = snapshot["camera"].value("autofocus_depth", m_slice_plane_z);
	}

	if (snapshot.contains("render_aabb_to_local")) from_json(snapshot.at("render_aabb_to_local"), m_render_aabb_to_local);
	m_render_aabb = snapshot.value("render_aabb", m_render_aabb);
	if (snapshot.contains("up_dir")) from_json(snapshot.at("up_dir"), m_up_dir);

	m_network_config = std::move(config);

	reset_network(false);

	m_training_step = m_network_config["snapshot"]["training_step"];
	m_loss_scalar.set(m_network_config["snapshot"]["loss"]);

	m_trainer->deserialize(m_network_config["snapshot"]);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		// If the snapshot appears to come from the same dataset as was already present
		// (or none was previously present, in which case it came from the snapshot
		// in the first place), load dataset-specific optimized quantities, such as
		// extrinsics, exposure, latents.
		if (snapshot["nerf"].contains("dataset") && m_nerf.training.dataset.is_same(snapshot["nerf"]["dataset"])) {
			if (snapshot["nerf"].contains("cam_pos_offset")) m_nerf.training.cam_pos_offset = snapshot["nerf"].at("cam_pos_offset").get<std::vector<AdamOptimizer<vec3>>>();
			if (snapshot["nerf"].contains("cam_rot_offset")) m_nerf.training.cam_rot_offset = snapshot["nerf"].at("cam_rot_offset").get<std::vector<RotationAdamOptimizer>>();
			if (snapshot["nerf"].contains("extra_dims_opt")) m_nerf.training.extra_dims_opt = snapshot["nerf"].at("extra_dims_opt").get<std::vector<VarAdamOptimizer>>();
			m_nerf.training.update_transforms();
			m_nerf.training.update_extra_dims();
		}
	}

	set_all_devices_dirty();
}

void Testbed::load_snapshot(const fs::path& path) {
	auto config = load_network_config(path);
	if (!config.contains("snapshot")) {
		throw std::runtime_error{fmt::format("File '{}' does not contain a snapshot.", path.str())};
	}

	load_snapshot(std::move(config));

	m_network_config_path = path;
}

void Testbed::load_snapshot(std::istream& stream, bool is_compressed) {
	auto config = load_network_config(stream, is_compressed);
	if (!config.contains("snapshot")) {
		throw std::runtime_error{"Given stream does not contain a snapshot."};
	}

	load_snapshot(std::move(config));

	// Network config path is unknown.
	m_network_config_path = "";
}

Testbed::CudaDevice::CudaDevice(int id, bool is_primary) : m_id{id}, m_is_primary{is_primary} {
	auto guard = device_guard();
	m_stream = std::make_unique<StreamAndEvent>();
	m_data = std::make_unique<Data>();
	m_render_worker = std::make_unique<ThreadPool>(is_primary ? 0u : 1u);
}

ScopeGuard Testbed::CudaDevice::device_guard() {
	int prev_device = cuda_device();
	if (prev_device == m_id) {
		return {};
	}

	set_cuda_device(m_id);
	return ScopeGuard{[prev_device]() {
		set_cuda_device(prev_device);
	}};
}

void Testbed::CudaDevice::set_network(const std::shared_ptr<Network<float, network_precision_t>>& network) {
	m_network = network;
}

void Testbed::CudaDevice::set_nerf_network(const std::shared_ptr<NerfNetwork<network_precision_t>>& nerf_network) {
	m_nerf_network = nerf_network;
	set_network(nerf_network);
}

void Testbed::sync_device(CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	if (!device.dirty()) {
		return;
	}

	if (device.is_primary()) {
		device.data().density_grid_bitfield_ptr = m_nerf.density_grid_bitfield.data();
		device.data().hidden_area_mask = render_buffer.hidden_area_mask();
		device.set_dirty(false);
		return;
	}

	m_stream.signal(device.stream());

	int active_device = cuda_device();
	auto guard = device.device_guard();

	device.data().density_grid_bitfield.resize(m_nerf.density_grid_bitfield.size());
	if (m_nerf.density_grid_bitfield.size() > 0) {
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(device.data().density_grid_bitfield.data(), device.id(), m_nerf.density_grid_bitfield.data(), active_device, m_nerf.density_grid_bitfield.bytes(), device.stream()));
	}

	device.data().density_grid_bitfield_ptr = device.data().density_grid_bitfield.data();

	if (m_network) {
		device.data().params.resize(m_network->n_params());
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(device.data().params.data(), device.id(), m_network->inference_params(), active_device, device.data().params.bytes(), device.stream()));
		device.nerf_network()->set_params(device.data().params.data(), device.data().params.data(), nullptr);
	}

	if (render_buffer.hidden_area_mask()) {
		auto ham = std::make_shared<Buffer2D<uint8_t>>(render_buffer.hidden_area_mask()->resolution());
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(ham->data(), device.id(), render_buffer.hidden_area_mask()->data(), active_device, ham->bytes(), device.stream()));
		device.data().hidden_area_mask = ham;
	} else {
		device.data().hidden_area_mask = nullptr;
	}

	device.set_dirty(false);
	device.signal(m_stream.get());
}

// From https://stackoverflow.com/questions/20843271/passing-a-non-copyable-closure-object-to-stdfunction-parameter
template <class F>
auto make_copyable_function(F&& f) {
	using dF = std::decay_t<F>;
	auto spf = std::make_shared<dF>(std::forward<F>(f));
	return [spf](auto&&... args) -> decltype(auto) {
		return (*spf)( decltype(args)(args)... );
	};
}

ScopeGuard Testbed::use_device(hipStream_t stream, CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	device.wait_for(stream);

	if (device.is_primary()) {
		device.set_render_buffer_view(render_buffer.view());
		return ScopeGuard{[&device, stream]() {
			device.set_render_buffer_view({});
			device.signal(stream);
		}};
	}

	int active_device = cuda_device();
	auto guard = device.device_guard();

	size_t n_pixels = product(render_buffer.in_resolution());

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<vec4, float>(device.stream(), &alloc, n_pixels, n_pixels);

	device.set_render_buffer_view({
		std::get<0>(scratch),
		std::get<1>(scratch),
		render_buffer.in_resolution(),
		render_buffer.spp(),
		device.data().hidden_area_mask,
	});

	return ScopeGuard{make_copyable_function([&render_buffer, &device, guard=std::move(guard), alloc=std::move(alloc), active_device, stream]() {
		// Copy device's render buffer's data onto the original render buffer
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(render_buffer.frame_buffer(), active_device, device.render_buffer_view().frame_buffer, device.id(), product(render_buffer.in_resolution()) * sizeof(vec4), device.stream()));
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(render_buffer.depth_buffer(), active_device, device.render_buffer_view().depth_buffer, device.id(), product(render_buffer.in_resolution()) * sizeof(float), device.stream()));

		device.set_render_buffer_view({});
		device.signal(stream);
	})};
}

void Testbed::set_all_devices_dirty() {
	for (auto& device : m_devices) {
		device.set_dirty(true);
	}
}

void Testbed::load_camera_path(const fs::path& path) {
	m_camera_path.load(path, mat4x3::identity());
}

}

