#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   python_api.cpp
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/thread_pool.h>

#include <json/json.hpp>

#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/eigen.h>
#include <pybind11/stl.h>
#include <pybind11_json/pybind11_json.hpp>

#include <filesystem/path.h>

#ifdef _WIN32
#include <GL/gl3w.h>            // Initialize with gl3wInit()
#else
#include <GL/glew.h>            // Initialize with glewInit()
#endif
#include <GLFW/glfw3.h>

using namespace tcnn;
using namespace Eigen;
using namespace nlohmann;
namespace py = pybind11;


NGP_NAMESPACE_BEGIN

void Testbed::override_sdf_training_data(py::array_t<float> points, py::array_t<float> distances) {
	py::buffer_info points_buf = points.request();
	py::buffer_info distances_buf = distances.request();

	if (points_buf.ndim != 2 || distances_buf.ndim != 1 || points_buf.shape[0] != distances_buf.shape[0] || points_buf.shape[1] != 3) {
		tlog::error() << "Invalid Points<->Distances data";
		return;
	}

	std::vector<Vector3f> points_cpu(points_buf.shape[0]);
	std::vector<float> distances_cpu(distances_buf.shape[0]);

	for (size_t i = 0; i < points_cpu.size(); ++i) {
		Vector3f pos = *((Vector3f*)points_buf.ptr + i);
		float dist = *((float*)distances_buf.ptr + i);

		pos = (pos - m_raw_aabb.min) / m_sdf.mesh_scale + 0.5f * (Vector3f::Ones() - (m_raw_aabb.max - m_raw_aabb.min) / m_sdf.mesh_scale);
		dist /= m_sdf.mesh_scale;

		points_cpu[i] = pos;
		distances_cpu[i] = dist;
	}

	CUDA_CHECK_THROW(hipMemcpyAsync(m_sdf.training.positions.data(), points_cpu.data(), points_buf.shape[0] * points_buf.shape[1] * sizeof(float), hipMemcpyHostToDevice, m_training_stream));
	CUDA_CHECK_THROW(hipMemcpyAsync(m_sdf.training.distances.data(), distances_cpu.data(), distances_buf.shape[0] * sizeof(float), hipMemcpyHostToDevice, m_training_stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(m_training_stream));
	m_sdf.training.size = points_buf.shape[0];
	m_sdf.training.idx = 0;
	m_sdf.training.max_size = m_sdf.training.size;
	m_sdf.training.generate_sdf_data_online = false;
}

py::array_t<float> Testbed::render_to_cpu(int width, int height, int spp, bool linear, float start_time, float end_time, float fps, float shutter_fraction) {
	m_windowless_render_surface.resize({width, height});
	m_windowless_render_surface.reset_accumulation();

	if (end_time < 0.f) {
		end_time = start_time;
	}

	auto start_cam_matrix = m_smoothed_camera;

	if (start_time >= 0.f) {
		set_camera_from_time(end_time);
		apply_camera_smoothing(1000.f / fps);
	} else {
		start_cam_matrix = m_smoothed_camera = m_camera;
	}

	auto end_cam_matrix = m_smoothed_camera;

	for (int i = 0; i < spp; ++i) {
		float start_alpha = ((float)i)/(float)spp * shutter_fraction;
		float end_alpha = ((float)i + 1.0f)/(float)spp * shutter_fraction;

		auto sample_start_cam_matrix = log_space_lerp(start_cam_matrix, end_cam_matrix, start_alpha);
		auto sample_end_cam_matrix = log_space_lerp(start_cam_matrix, end_cam_matrix, end_alpha);

		if (start_time >= 0.f) {
			set_camera_from_time(start_time + (end_time-start_time) * (start_alpha + end_alpha) / 2.0f);
			m_smoothed_camera = m_camera;
		}

		if (m_autofocus) {
			autofocus();
		}

		render_frame(sample_start_cam_matrix, sample_end_cam_matrix, m_windowless_render_surface, !linear);
	}

	// For cam smoothing when rendering the next frame.
	m_smoothed_camera = end_cam_matrix;

	py::array_t<float> result({height, width, 4});
	py::buffer_info buf = result.request();

	CUDA_CHECK_THROW(hipMemcpy2DFromArray(buf.ptr, width * sizeof(float) * 4, m_windowless_render_surface.surface_provider().array(), 0, 0, width * sizeof(float) * 4, height, hipMemcpyDeviceToHost));
	return result;
}

py::array_t<float> Testbed::screenshot(bool linear) const {
	std::vector<float> tmp(m_window_res.prod() * 4);
	glReadPixels(0, 0, m_window_res.x(), m_window_res.y(), GL_RGBA, GL_FLOAT, tmp.data());

	py::array_t<float> result({m_window_res.y(), m_window_res.x(), 4});
	py::buffer_info buf = result.request();
	float* data = (float*)buf.ptr;

	// Linear, alpha premultiplied, Y flipped
	ThreadPool pool;
	pool.parallelFor<size_t>(0, m_window_res.y(), [&](size_t y) {
		size_t base = y * m_window_res.x();
		size_t base_reverse = (m_window_res.y() - y - 1) * m_window_res.x();
		for (uint32_t x = 0; x < m_window_res.x(); ++x) {
			size_t px = base + x;
			size_t px_reverse = base_reverse + x;
			data[px_reverse*4+0] = linear ? srgb_to_linear(tmp[px*4+0]) : tmp[px*4+0];
			data[px_reverse*4+1] = linear ? srgb_to_linear(tmp[px*4+1]) : tmp[px*4+1];
			data[px_reverse*4+2] = linear ? srgb_to_linear(tmp[px*4+2]) : tmp[px*4+2];
			data[px_reverse*4+3] = tmp[px*4+3];
		}
	});

	return result;
}

//TODO: use this when magic_enum starts working with CUDA
// template <typename E, typename T>
// void register_enum(T& parent) {
// 	py::enum_<E> e(parent, std::string{magic_enum::enum_type_name<E>()}.c_str());
// 	for (auto [val, name] : magic_enum::enum_entries<E>()) {
// 		e.value(std::string{name}.c_str(), val);
// 	}
// 	e.export_values();
// }

PYBIND11_MODULE(pyngp, m) {
	m.doc() = "Instant neural graphics primitives";

	py::enum_<ETestbedMode>(m, "TestbedMode")
		.value("Nerf", ETestbedMode::Nerf)
		.value("Sdf", ETestbedMode::Sdf)
		.value("Image", ETestbedMode::Image)
		.value("Volume", ETestbedMode::Volume)
		.export_values();

	py::enum_<ERenderMode>(m, "RenderMode")
		.value("AO", ERenderMode::AO)
		.value("Shade", ERenderMode::Shade)
		.value("Normals", ERenderMode::Normals)
		.value("Positions", ERenderMode::Positions)
		.value("Depth", ERenderMode::Depth)
		.value("Distance", ERenderMode::Distance)
		.value("Stepsize", ERenderMode::Stepsize)
		.value("Distortion", ERenderMode::Distortion)
		.value("Cost", ERenderMode::Cost)
		.value("Slice", ERenderMode::Slice)
		.export_values();

	py::enum_<ERandomMode>(m, "RandomMode")
		.value("Random", ERandomMode::Random)
		.value("Halton", ERandomMode::Halton)
		.value("Sobol", ERandomMode::Sobol)
		.value("Stratified", ERandomMode::Stratified)
		.export_values();

	py::enum_<ELossType>(m, "LossType")
		.value("L2", ELossType::L2)
		.value("L1", ELossType::L1)
		.value("Mape", ELossType::Mape)
		.value("Smape", ELossType::Smape)
		.value("SmoothL1", ELossType::SmoothL1)
		.value("LogL1", ELossType::LogL1)
		.value("RelativeL2", ELossType::RelativeL2)
		.export_values();

	py::enum_<ENerfActivation>(m, "NerfActivation")
		.value("None", ENerfActivation::None)
		.value("ReLU", ENerfActivation::ReLU)
		.value("Logistic", ENerfActivation::Logistic)
		.value("Exponential", ENerfActivation::Exponential)
		.export_values();

	py::enum_<EMeshSdfMode>(m, "MeshSdfMode")
		.value("Watertight", EMeshSdfMode::Watertight)
		.value("Raystab", EMeshSdfMode::Raystab)
		.value("PathEscape", EMeshSdfMode::PathEscape)
		.export_values();

	py::enum_<EColorSpace>(m, "ColorSpace")
		.value("Linear", EColorSpace::Linear)
		.value("SRGB", EColorSpace::SRGB)
		.export_values();

	py::enum_<ETonemapCurve>(m, "TonemapCurve")
		.value("Identity", ETonemapCurve::Identity)
		.value("ACES", ETonemapCurve::ACES)
		.value("Hable", ETonemapCurve::Hable)
		.value("Reinhard", ETonemapCurve::Reinhard)
		.export_values();

	py::class_<Testbed> testbed(m, "Testbed");
	testbed
		.def(py::init<ETestbedMode>())
		.def(py::init<ETestbedMode, const std::string&, const std::string&>())
		.def(py::init<ETestbedMode, const std::string&, const json&>())
		.def("load_training_data", &Testbed::load_training_data, "Load training data from a given path.")
		.def("clear_training_data", &Testbed::clear_training_data, "Clears training data to free up GPU memory.")
		// General control
		.def("init_window", &Testbed::init_window, "Init a GLFW window that shows real-time progress and a GUI.", py::arg("width"), py::arg("height"), py::arg("hidden")=false)
		.def("want_repl", &Testbed::want_repl, "returns true if the user clicked the 'I want a repl' button")
		.def("frame", &Testbed::frame, "Process a single frame. Renders if a window was previously created.")
		.def("render", &Testbed::render_to_cpu, "Renders an image at the requested resolution. Does not require a window.",
			py::arg("width")=1920, py::arg("height")=1080, py::arg("spp")=1, py::arg("linear")=true,
			py::arg("start_t")=-1.f, py::arg("end_t")=-1.f,
			py::arg("fps")=30.f, py::arg("shutter_fraction")=1.0f)
		.def("screenshot", &Testbed::screenshot, "Takes a screenshot of the current window contents.", py::arg("linear")=true)
		// TODO: revisit this binding and return the mesh a python array rather than the number of triangles
		// .def("marching_cubes", &Testbed::marching_cubes, py::arg("path"), py::arg("res")=128, py::arg("thresh")=2.f, py::arg("unwrap")=false, "Runs marching cubes at the requested res and outputs an OBJ to the given path. Does not require a window.")
		.def("destroy_window", &Testbed::destroy_window, "Destroy the window again.")
		.def("train", &Testbed::train, "Perform a specified number of training steps.")
		.def("reset", &Testbed::reset_network, "Reset training.")
		.def("reset_accumulation", &Testbed::reset_accumulation, "Reset rendering accumulation.")
		.def("reload_network_from_file", &Testbed::reload_network_from_file, py::arg("path")="", "Reload the network from a config file.")
		.def("reload_network_from_json", &Testbed::reload_network_from_json, "Reload the network from a json object.")
		.def("override_sdf_training_data", &Testbed::override_sdf_training_data, "Override the training data for learning a signed distance function")
		.def("calculate_iou", &Testbed::calculate_iou, "Calculate the intersection over union error value",
			py::arg("n_samples")=128*1024*1024, py::arg("scale_existing_results_factor")=0.0f, py::arg("blocking")=true, py::arg("force_use_octree")=true)
		.def("n_params", &Testbed::n_params, "get number of trainable parameters")
		.def("n_encoding_params", &Testbed::n_encoding_params, "get number of trainable parameters in the encoding")
		.def("save_snapshot", &Testbed::save_snapshot, py::arg("path"), py::arg("include_optimizer_state")=false, "Save a snapshot of the currently trained model")
		.def("load_snapshot", &Testbed::load_snapshot, py::arg("path"), "Load a previously saved snapshot")
		.def("load_camera_path", &Testbed::load_camera_path, "Load a camera path", py::arg("path"))
		;

	// Interesting members.
	testbed
		.def_readwrite("dynamic_res", &Testbed::m_dynamic_res)
		.def_readwrite("fixed_res_factor", &Testbed::m_fixed_res_factor)
		.def_readwrite("background_color", &Testbed::m_background_color)
		.def_readwrite("shall_train", &Testbed::m_train)
		.def_readwrite("shall_train_encoding", &Testbed::m_train_encoding)
		.def_readwrite("shall_train_network", &Testbed::m_train_network)
		.def_readwrite("render_groundtruth", &Testbed::m_render_ground_truth)
		.def_readwrite("render_mode", &Testbed::m_render_mode)
		.def_readwrite("slice_plane_z", &Testbed::m_slice_plane_z)
		.def_readwrite("dof", &Testbed::m_dof)
		.def_readwrite("autofocus", &Testbed::m_autofocus)
		.def_readwrite("autofocus_target", &Testbed::m_autofocus_target)
		.def_readwrite("floor_enable", &Testbed::m_floor_enable)
		.def_readwrite("exposure", &Testbed::m_exposure)
		.def_property("scale", &Testbed::scale, &Testbed::set_scale)
		.def_readonly("bounding_radius", &Testbed::m_bounding_radius)
		.def_readwrite("render_aabb", &Testbed::m_render_aabb)
		.def_property("fov", &Testbed::fov, &Testbed::set_fov)
		.def_property("fov_xy", &Testbed::fov_xy, &Testbed::set_fov_xy)
		.def_readwrite("fov_axis", &Testbed::m_fov_axis)
		.def_readwrite("zoom", &Testbed::m_zoom)
		.def_readwrite("screen_center", &Testbed::m_screen_center)
		.def("set_nerf_camera_matrix", &Testbed::set_nerf_camera_matrix)
		.def("set_camera_to_training_view", &Testbed::set_camera_to_training_view)
		.def("compute_image_mse", &Testbed::compute_image_mse)
		.def_readwrite("camera_matrix", &Testbed::m_camera)
		.def_readwrite("up_dir", &Testbed::m_up_dir)
		.def_readwrite("sun_dir", &Testbed::m_sun_dir)
		.def_property("look_at", &Testbed::look_at, &Testbed::set_look_at)
		.def_property("view_dir", &Testbed::view_dir, &Testbed::set_view_dir)
		.def_readwrite("max_level_rand_training", &Testbed::m_max_level_rand_training)
		.def_readwrite("visualized_dimension", &Testbed::m_visualized_dimension)
		.def_readwrite("visualized_layer", &Testbed::m_visualized_layer)
		.def_readonly("loss", &Testbed::m_loss_scalar)
		.def_readonly("training_step", &Testbed::m_training_step)
		.def_readonly("nerf", &Testbed::m_nerf)
		.def_readonly("sdf", &Testbed::m_sdf)
		.def_readonly("image", &Testbed::m_image)
		.def_readwrite("camera_smoothing", &Testbed::m_camera_smoothing)
		.def_readwrite("display_gui", &Testbed::m_imgui_enabled)
		.def_readwrite("visualize_unit_cube", &Testbed::m_visualize_unit_cube)
		.def_readwrite("snap_to_pixel_centers", &Testbed::m_snap_to_pixel_centers)
		.def_readwrite("color_space", &Testbed::m_color_space)
		.def_readwrite("tonemap_curve", &Testbed::m_tonemap_curve)
		;

	py::class_<Testbed::Nerf> nerf(testbed, "Nerf");
	nerf
		.def_readonly("training", &Testbed::Nerf::training)
		.def_readwrite("rgb_activation", &Testbed::Nerf::rgb_activation)
		.def_readwrite("density_activation", &Testbed::Nerf::density_activation)
		.def_readwrite("sharpen", &Testbed::Nerf::sharpen)
		.def_readwrite("render_with_camera_distortion", &Testbed::Nerf::render_with_camera_distortion)
		.def_readwrite("rendering_min_alpha", &Testbed::Nerf::rendering_min_alpha)
		.def_readwrite("cone_angle_constant", &Testbed::Nerf::cone_angle_constant)
		.def_readwrite("visualize_cameras", &Testbed::Nerf::visualize_cameras)
		;

	py::class_<BoundingBox> boundingbox(m, "BoundingBox");
	boundingbox
		.def_readwrite("min", &BoundingBox::min)
		.def_readwrite("max", &BoundingBox::max)
		;

	py::class_<BRDFParams> brdfparams(m, "BRDFParams");
	brdfparams
		.def_readwrite("metallic", &BRDFParams::metallic)
		.def_readwrite("subsurface", &BRDFParams::subsurface)
		.def_readwrite("specular", &BRDFParams::specular)
		.def_readwrite("roughness", &BRDFParams::roughness)
		.def_readwrite("sheen", &BRDFParams::sheen)
		.def_readwrite("clearcoat", &BRDFParams::clearcoat)
		.def_readwrite("clearcoat_gloss", &BRDFParams::clearcoat_gloss)
		.def_readwrite("basecolor", &BRDFParams::basecolor)
		.def_readwrite("ambientcolor", &BRDFParams::ambientcolor)
		;

	py::class_<NerfDataset> nerfdataset(m, "NerfDataset");
	nerfdataset
		.def_readonly("focal_lengths", &NerfDataset::focal_lengths)
		.def_readonly("transforms", &NerfDataset::xforms)
		.def_readonly("render_aabb", &NerfDataset::render_aabb)
		.def_readonly("up", &NerfDataset::up)
		.def_readonly("offset", &NerfDataset::offset)
		.def_readonly("n_images", &NerfDataset::n_images)
		.def_readonly("image_resolution", &NerfDataset::image_resolution)
		.def_readonly("envmap_resolution", &NerfDataset::envmap_resolution)
		.def_readonly("scale", &NerfDataset::scale)
		.def_readonly("aabb_scale", &NerfDataset::aabb_scale)
		.def_readonly("principal_point", &NerfDataset::principal_point)
		.def_readonly("from_mitsuba", &NerfDataset::from_mitsuba)
		.def_readonly("is_hdr", &NerfDataset::is_hdr)
		;

	py::class_<Testbed::Nerf::Training>(nerf, "Training")
		.def_readwrite("random_bg_color", &Testbed::Nerf::Training::random_bg_color)
		.def_readwrite("linear_colors", &Testbed::Nerf::Training::linear_colors)
		.def_readwrite("loss_type", &Testbed::Nerf::Training::loss_type)
		.def_readwrite("snap_to_pixel_centers", &Testbed::Nerf::Training::snap_to_pixel_centers)
		.def_readwrite("optimize_extrinsics", &Testbed::Nerf::Training::optimize_extrinsics)
		.def_readwrite("optimize_exposure", &Testbed::Nerf::Training::optimize_exposure)
		.def_readwrite("optimize_distortion", &Testbed::Nerf::Training::optimize_distortion)
		.def_readwrite("optimize_focal_length", &Testbed::Nerf::Training::optimize_focal_length)
		.def_readwrite("n_steps_between_cam_updates", &Testbed::Nerf::Training::n_steps_between_cam_updates)
		.def_readwrite("sample_focal_plane_proportional_to_error", &Testbed::Nerf::Training::sample_focal_plane_proportional_to_error)
		.def_readwrite("sample_image_proportional_to_error", &Testbed::Nerf::Training::sample_image_proportional_to_error)
		.def_readwrite("include_sharpness_in_error", &Testbed::Nerf::Training::include_sharpness_in_error)
		.def_readonly("transforms", &Testbed::Nerf::Training::transforms)
		.def_readonly("focal_lengths", &Testbed::Nerf::Training::focal_lengths)
		.def_readonly("image_resolution", &Testbed::Nerf::Training::image_resolution)
		.def_readwrite("density_grid_decay", &Testbed::Nerf::Training::density_grid_decay)
		.def_readonly("dataset", &Testbed::Nerf::Training::dataset)
		;

	py::class_<Testbed::Sdf> sdf(testbed, "Sdf");
	sdf
		.def_readonly("training", &Testbed::Sdf::training)
		.def_readwrite("mesh_sdf_mode", &Testbed::Sdf::mesh_sdf_mode)
		.def_readwrite("analytic_normals", &Testbed::Sdf::analytic_normals)
		.def_readwrite("shadow_sharpness", &Testbed::Sdf::shadow_sharpness)
		.def_readwrite("fd_normals_epsilon", &Testbed::Sdf::fd_normals_epsilon)
		.def_readwrite("use_triangle_octree", &Testbed::Sdf::use_triangle_octree)
		.def_readwrite("zero_offset", &Testbed::Sdf::zero_offset)
		.def_readwrite("distance_scale", &Testbed::Sdf::distance_scale)
		.def_readwrite("calculate_iou_online", &Testbed::Sdf::calculate_iou_online)
		.def_readwrite("groundtruth_spheremarch", &Testbed::Sdf::groundtruth_spheremarch)
		.def_readwrite("brdf", &Testbed::Sdf::brdf)
		;

	py::class_<Testbed::Sdf::Training>(sdf, "Training")
		.def_readwrite("generate_sdf_data_online", &Testbed::Sdf::Training::generate_sdf_data_online)
		;

	py::class_<Testbed::Image> image(testbed, "Image");
	image
		.def_readonly("training", &Testbed::Image::training)
		.def_readwrite("random_mode", &Testbed::Image::random_mode)
		.def_readwrite("pos", &Testbed::Image::pos)
		;

	py::class_<Testbed::Image::Training>(image, "Training")
		.def_readwrite("snap_to_pixel_centers", &Testbed::Image::Training::snap_to_pixel_centers)
		.def_readwrite("linear_colors", &Testbed::Image::Training::linear_colors)
		;
}

NGP_NAMESPACE_END
